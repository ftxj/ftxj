#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main() {
  int * input;
  int * output;
  int * input_d;
  int * output_d;

  input  = (int*)malloc(2*sizeof(int));
  output = (int*)malloc(2*sizeof(int));

  input[0] = 10;
  input[1] = 20;
  output[0] = 0;
  output[1] = 0;

	cout << "Before the copy kernel." << endl;
	cout << "Input value:\t" << input[0] << "\t" << input[1] << endl;
  cout << "Output value:\t" << output[0] << "\t" << output[1] << endl;

  hipMalloc((void**)&input_d,  2*sizeof(int));
  hipMalloc((void**)&output_d, 2*sizeof(int));

  hipMemcpy(input_d, input, 2*sizeof(int), hipMemcpyHostToDevice);
  
  hipModule_t module;
  hipFunction_t kernel;

  hipModuleLoad(&module, "copy.cubin");
  hipModuleGetFunction(&kernel, module, "kern");

  void * args[2] = {&input_d, &output_d};
  hipModuleLaunchKernel(kernel, 1, 1, 1, 
                 1, 1, 1, 
                 0, 0, args, 0);
  hipDeviceSynchronize();

  hipMemcpy(output, output_d, 2*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(input, input_d, 2*sizeof(int), hipMemcpyDeviceToHost);

	cout << "After the copy kernel." << endl;
	cout << "Input value:\t" << input[0] << "\t" << input[1] << endl;
  cout << "Output value:\t" << output[0] << "\t" << output[1] << endl;


  return 0;
  
}
