#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

char* concat(const char *s1, const char *s2)
{
    char *result = (char*)malloc(strlen(s1) + strlen(s2) + 1); // +1 for the null-terminator
    // in real code you would check for errors in malloc here
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void run(char * name){
	char * file_name = concat(name, ".cubin");

	int *output;
	hipMalloc((void**)&output, sizeof(int)*128);

	hipModule_t module;
	hipFunction_t kernel;

	hipModuleLoad(&module, file_name);
	hipModuleGetFunction(&kernel, module, "kern");

	void * args[1] = {&output};
	hipModuleLaunchKernel(kernel, 1, 1, 1,
			32, 1, 1,
			32*1024, 0, args, 0);

	int *output_h = (int*)malloc(sizeof(int)*32);

	hipMemcpy(output_h, output, sizeof(int)*32, hipMemcpyDeviceToHost);

	printf("%s took %d clocks.\n", name, output_h[0]);
	printf("Each instruction takes %.2f clocks.\n\n", (float)output_h[0]/(128.0*128.0));

	hipFree(output);
	free(output_h);
}

int main(){
	run("lds32");
	return 0;
}
